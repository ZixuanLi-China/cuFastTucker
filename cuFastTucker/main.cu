#include "hip/hip_runtime.h"
#include <stdio.h>
#include "tools.h"
#include "kernel.h"

using namespace std;

int iter_number;

type_of_data learn_alpha_a;
type_of_data learn_beta_a;
type_of_data lambda_a;

type_of_data learn_alpha_b;
type_of_data learn_beta_b;
type_of_data lambda_b;

type_of_data learn_rate_a;
type_of_data learn_rate_b;

type_of_data train_rmse;
type_of_data train_mae;

type_of_data test_rmse;
type_of_data test_mae;

type_of_data best_train_rmse;
type_of_data best_train_mae;

type_of_data best_test_rmse;
type_of_data best_test_mae;

char *InputPath_train;
char *InputPath_test;

int order;
int core_kernel;
int core_length;

int *dimen;
int core_dimen;

double data_norm;

int nnz_train;
type_of_data **value_train_host;
type_of_data **value_train_device;
type_of_data **value_train_host_to_device;
int **index_train_host;
int **index_train_device;
int **index_train_host_to_device;

int nnz_test;
type_of_data *value_test_host;
type_of_data *value_test_device;
int *index_test_host;
int *index_test_device;

type_of_data **parameter_a_host;
type_of_data **parameter_a_host_to_device;
type_of_data **parameter_a_device;

type_of_data **parameter_b_host;
type_of_data **parameter_b_host_to_device;
type_of_data **parameter_b_device;

double time_spend = 0.0;
double start_time;
double mid_time;
double stop_time;

int main(int argc, char *argv[]) {

	if (argc == 13) {

		InputPath_train = argv[1];
		InputPath_test = argv[2];

		core_kernel = atoi(argv[3]);
		order = atoi(argv[4]);
		core_dimen = atoi(argv[5]);

		iter_number = atoi(argv[6]);

		learn_alpha_a = atof(argv[7]);
		learn_beta_a = atof(argv[8]);
		lambda_a = atof(argv[9]);

		learn_alpha_b = atof(argv[10]);
		learn_beta_b = atof(argv[11]);
		lambda_b = atof(argv[12]);

		core_length = 1;
		for (int i = 0; i < order; i++) {
			core_length *= core_dimen;
		}

	}

	printf("learn_alpha_a:%f\tlearn_beta_a:%f\tlambda_a:%f\n", learn_alpha_a,
			learn_beta_a, lambda_a);
	printf("learn_alpha_b:%f\tlearn_beta_b:%f\tlambda_b:%f\n", learn_alpha_b,
			learn_beta_b, lambda_b);

	Getting_Input(InputPath_train, InputPath_test, order, &dimen, &nnz_train,
			&nnz_test, &index_train_host, &value_train_host, &index_test_host,
			&value_test_host, &data_norm);

	printf("nnz_train:\t%d\n", nnz_train);
	printf("nnz_test:\t%d\n", nnz_test);

	for (int i = 0; i < order; i++) {
		printf("order %d:\t%d\n", i + 1, dimen[i]);
	}
	printf("data_norm:\t%f\n", data_norm);

	Parameter_Initialization(order, core_kernel, core_length, core_dimen, dimen,
			data_norm, &parameter_a_host, &parameter_b_host);

	Cuda_Parameter_Initialization(order, core_kernel, core_dimen, dimen,
			nnz_train, value_train_host, &value_train_device,
			&value_train_host_to_device, index_train_host, &index_train_device,
			&index_train_host_to_device, nnz_test, value_test_host,
			&value_test_device, index_test_host, &index_test_device,
			parameter_a_host, parameter_b_host, &parameter_a_device,
			&parameter_b_device, &parameter_a_host_to_device,
			&parameter_b_host_to_device);

	GET_RMSE_AND_MAE(order, core_kernel, core_dimen, parameter_a_device,
			parameter_b_device, nnz_train, value_train_host_to_device,
			index_train_host_to_device, &best_train_rmse, &best_train_mae);

	GET_RMSE_AND_MAE(order, core_kernel, core_dimen, parameter_a_device,
			parameter_b_device, nnz_test, value_test_device, index_test_device,
			&best_test_rmse, &best_test_mae);

	printf(
			"initial:\ttrain rmse:%f\ttest rmse:%f\ttrain mae:%f\ttest mae:%f\t\n",
			best_train_rmse, best_test_rmse, best_train_mae, best_test_mae);
	printf(
			"iter\ttrain rmse\ttest rmse\ttrain mae\ttest mae\tfactor time\tcore time\ttotal time\tcumulative time\n");

	for (int i = 0; i < iter_number; i++) {

		learn_rate_a = learn_alpha_a / (1 + learn_beta_a * pow(i, 1.5));
		learn_rate_b = learn_alpha_b / (1 + learn_beta_b * pow(i, 1.5));

		start_time = Seconds();

		Update_Parameter_A(order, core_kernel, core_dimen, parameter_a_device,
				parameter_b_device, nnz_train, value_train_host_to_device,
				index_train_host_to_device, learn_rate_a, lambda_a);

		mid_time = Seconds();

		Update_Parameter_B_Batch(order, core_kernel, core_dimen,
				parameter_a_device, parameter_b_device, nnz_train,
				value_train_host_to_device, index_train_host_to_device,
				learn_rate_b, lambda_b);

		stop_time = Seconds();
		time_spend += stop_time - start_time;

		GET_RMSE_AND_MAE(order, core_kernel, core_dimen, parameter_a_device,
				parameter_b_device, nnz_train, value_train_host_to_device,
				index_train_host_to_device, &train_rmse, &train_mae);

		GET_RMSE_AND_MAE(order, core_kernel, core_dimen, parameter_a_device,
				parameter_b_device, nnz_test, value_test_device,
				index_test_device, &test_rmse, &test_mae);

		Select_Best_Result(&train_rmse, &train_mae, &test_rmse, &test_mae,
				&best_train_rmse, &best_train_mae, &best_test_rmse,
				&best_test_mae);

		printf("%d\t%f\t%f\t%f\t%f\t%f\t%f\t%f\t%f\n", i, train_rmse, test_rmse,
				train_mae, test_mae, mid_time - start_time,
				stop_time - mid_time, stop_time - start_time, time_spend);
	}

	printf("best:\ttrain rmse:%f\ttest rmse:%f\ttrain mae:%f\ttest mae:%f\t\n",
			best_train_rmse, best_test_rmse, best_train_mae, best_test_mae);

	return 0;
}
