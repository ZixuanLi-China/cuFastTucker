#include "hip/hip_runtime.h"
#include "kernel.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>

#define type_of_data float
#define grid_size 1024*1024
#define block_size 128
#define data_part 1
#define warp_size 32
#define sum_size 1024
#define error_size 1024
#define sm 1024

#define mask 0xffffffff

using namespace std;

__global__ void Structure_Core_Tensor(const int order, const int core_length,
		const int core_dimen, type_of_data *parameter_g, type_of_data **g) {

	int core = core_dimen;
	int worker = block_size / core;
	int lane_id = threadIdx.x % core;
	int local_id = threadIdx.x / core;
	int worker_id = worker * blockIdx.x + local_id;
	int workers = worker * gridDim.x;

	for (int order_index = 0; order_index < order; order_index++) {
		int length = core_length / core_dimen;
		for (int index = worker_id; index < length; index += workers) {

			int g_index = 0;
			int weight = core_length;
			int parameter_b_index = index;
			for (int inner_order_index = 0; inner_order_index < order;
					inner_order_index++) {
				weight /= core_dimen;
				if (inner_order_index != order_index) {
					g_index += (parameter_b_index % core_dimen) * weight;
					parameter_b_index /= core_dimen;
				} else {
					g_index += lane_id * weight;
				}
			}
			g[order_index][index * core_dimen + lane_id] = parameter_g[g_index];
		}
	}
}

__global__ void Update_Parameter_A_SGD(const int order, const int core_length,
		const int core_dimen, type_of_data **parameter_a, const int nnz,
		type_of_data *value, int *index, type_of_data **g,
		const type_of_data learn_rate_a, const type_of_data lambda_a,
		const int update_order) {

	int core = core_dimen;
	int worker = block_size / core;
	int local = error_size / core;
	int lane_id = threadIdx.x % core;
	int local_id = threadIdx.x / core;
	int worker_id = worker * blockIdx.x + local_id;
	int workers = worker * gridDim.x;

	for (int nnz_index = worker_id; nnz_index < nnz; nnz_index += workers) {
		int order_index = nnz_index * order;
		type_of_data gs = 0.0;
		for (int g_index = 0; g_index < core_length / core_dimen; g_index++) {
			int parameter_a_index = g_index;
			type_of_data s = 1.0;
			for (int inner_order_index = 0; inner_order_index < order;
					inner_order_index++) {
				if (inner_order_index != update_order) {
					s *= parameter_a[inner_order_index][index[order_index
							+ inner_order_index] * core_dimen
							+ parameter_a_index % core_dimen];
					parameter_a_index /= core_dimen;
				}

			}
			gs += s * g[update_order][g_index * core_dimen + lane_id];
		}

		type_of_data p_a_temp = parameter_a[update_order][index[order_index
				+ update_order] * core_dimen + lane_id];

		type_of_data p_a_gs = p_a_temp * gs;

		int temp = core;
		while (temp != 1) {
			temp /= 2;
			p_a_gs += __shfl_down_sync(mask, p_a_gs, temp);
		}

		p_a_gs = __shfl_sync(mask, p_a_gs, (local_id % local) * core);

		parameter_a[update_order][index[order_index + update_order] * core_dimen
				+ lane_id] -=
				learn_rate_a
						* (-value[nnz_index] * gs + p_a_gs * gs
								+ lambda_a
										* parameter_a[update_order][index[order_index
												+ update_order] * core_dimen
												+ lane_id]);

	}
}

void Update_Parameter_A(const int order, const int core_length,
		const int core_dimen, type_of_data **parameter_a, const int nnz,
		type_of_data **value, int **index, type_of_data **g,
		const type_of_data learn_rate_a, const type_of_data lambda_a) {

	int data_per_part = nnz / data_part + 1;
	for (int update_order = 0; update_order < order; update_order++) {
		for (int i = 0; i < data_part - 1; i++) {
			Update_Parameter_A_SGD<<<grid_size,
			block_size>>>(order, core_length, core_dimen, parameter_a,
					data_per_part, value[i], index[i], g, learn_rate_a,
					lambda_a, update_order);
			hipDeviceSynchronize();
		}

		Update_Parameter_A_SGD<<<grid_size,
		block_size>>>(order, core_length, core_dimen, parameter_a,
				nnz - (data_part - 1) * data_per_part, value[data_part - 1],
				index[data_part - 1], g, learn_rate_a, lambda_a, update_order);
		hipDeviceSynchronize();

	}

}

__global__ void Update_Parameter_G_Gradient(const int order,
		const int core_length, const int core_dimen, type_of_data **parameter_a,
		type_of_data *parameter_g, const int nnz, const type_of_data *value,
		const int *index, type_of_data *g_sum) {

	int core = core_dimen;
	int worker = block_size / core;
	int local = error_size / core;
	int lane_id = threadIdx.x % core;
	int local_id = threadIdx.x / core;
	int worker_id = worker * blockIdx.x + local_id;
	int workers = worker * gridDim.x;

	extern __shared__ type_of_data h_shared[];

	for (int nnz_index = worker_id; nnz_index < nnz; nnz_index += workers) {
		int order_index = nnz_index * order;
		type_of_data x_r = 0.0;
		for (int g_index = 0; g_index < core_length / core_dimen; g_index++) {
			int parameter_a_and_b_index = g_index;
			type_of_data h = parameter_a[0][index[order_index] * core_dimen
					+ lane_id];
			for (int inner_order_index = 0; inner_order_index < order;
					inner_order_index++) {
				if (inner_order_index != 0) {
					h *= parameter_a[inner_order_index][index[order_index
							+ inner_order_index] * core_dimen
							+ parameter_a_and_b_index % core_dimen];
					parameter_a_and_b_index /= core_dimen;
				}
			}
			h_shared[g_index * block_size + threadIdx.x] = h;
			x_r += h * parameter_g[g_index * core_dimen + lane_id];
		}

		int temp = core;
		while (temp != 1) {
			temp /= 2;
			x_r += __shfl_down_sync(mask, x_r, temp);
		}

		x_r = __shfl_sync(mask, x_r, (local_id % local) * core);

		x_r -= value[nnz_index];

		for (int g_index = 0; g_index < core_length / core_dimen; g_index++) {
			atomicAdd(
					&g_sum[(nnz_index % sum_size) * core_length
							+ g_index * core_dimen + lane_id],
					x_r * h_shared[g_index * block_size + threadIdx.x]);
		}

	}
}

__global__ void Update_Parameter_G_Gradient_Gobal(const int order,
		const int core_length, const int core_dimen, type_of_data **parameter_a,
		type_of_data *parameter_g, const int nnz, const type_of_data *value,
		const int *index, type_of_data *g_sum,
		type_of_data *h_shared) {

	int core = core_dimen;
	int worker = block_size / core;
	int local = error_size / core;
	int lane_id = threadIdx.x % core;
	int local_id = threadIdx.x / core;
	int worker_id = worker * blockIdx.x + local_id;
	int workers = worker * gridDim.x;

	for (int nnz_index = worker_id; nnz_index < nnz; nnz_index += workers) {
		int order_index = nnz_index * order;
		type_of_data x_r = 0.0;
		for (int g_index = 0; g_index < core_length / core_dimen; g_index++) {
			int parameter_a_and_b_index = g_index;
			type_of_data h = parameter_a[0][index[order_index] * core_dimen
					+ lane_id];
			for (int inner_order_index = 0; inner_order_index < order;
					inner_order_index++) {
				if (inner_order_index != 0) {
					h *= parameter_a[inner_order_index][index[order_index
							+ inner_order_index] * core_dimen
							+ parameter_a_and_b_index % core_dimen];
					parameter_a_and_b_index /= core_dimen;
				}
			}
			h_shared[(nnz_index % sm) * g_index * core_dimen + lane_id] = h;
			x_r += h * parameter_g[g_index * core_dimen + lane_id];
		}

		int temp = core;
		while (temp != 1) {
			temp /= 2;
			x_r += __shfl_down_sync(mask, x_r, temp);
		}

		x_r = __shfl_sync(mask, x_r, (local_id % local) * core);

		x_r -= value[nnz_index];

		for (int g_index = 0; g_index < core_length / core_dimen; g_index++) {
			atomicAdd(
					&g_sum[(nnz_index % sum_size) * core_length
							+ g_index * core_dimen + lane_id],
					x_r
							* h_shared[(nnz_index % sm) * g_index * core_dimen
									+ lane_id]);
		}

	}
}

__global__ void Parameter_G_Gradient_Sum(const int order, const int core_length,
		const int core_dimen, const int nnz,
		type_of_data *g_sum, type_of_data *g_grad) {

	int core = core_dimen;
	int worker = block_size / core;
	int lane_id = threadIdx.x % core;
	int local_id = threadIdx.x / core;
	int worker_id = worker * blockIdx.x + local_id;
	int workers = worker * gridDim.x;

	for (int g_index = worker_id; g_index < core_length / core_dimen; g_index +=
			workers) {
		for (int sum_size_index = 0; sum_size_index < sum_size;
				sum_size_index++) {
			g_grad[g_index * core_dimen + lane_id] += g_sum[sum_size_index
					* core_length + g_index * core_dimen + lane_id];
		}
		g_grad[g_index * core_dimen + lane_id] /= nnz;
	}
}

__global__ void Update_Parameter_G(const int order, const int core_length,
		const int core_dimen, type_of_data *parameter_g, type_of_data *g_grad,
		const type_of_data learn_rate_g, const type_of_data lambda_g) {

	int core = core_dimen;
	int worker = block_size / core;
	int lane_id = threadIdx.x % core;
	int local_id = threadIdx.x / core;
	int worker_id = worker * blockIdx.x + local_id;
	int workers = worker * gridDim.x;

	for (int g_index = worker_id; g_index < core_length / core_dimen; g_index +=
			workers) {
		parameter_g[g_index * core_dimen + lane_id] -=
				learn_rate_g
						* (g_grad[g_index * core_dimen + lane_id]
								+ lambda_g
										* parameter_g[g_index * core_dimen
												+ lane_id]);

	}
}

void Update_Parameter_G_Batch(const int order, const int core_length,
		const int core_dimen, type_of_data **parameter_a,
		type_of_data *parameter_g, const int nnz, type_of_data **value,
		int **index, const type_of_data learn_rate_g,
		const type_of_data lambda_g, int model) {

	type_of_data *g_sum;
	type_of_data *g_grad;
	hipMalloc((void**) &g_sum, sum_size * core_length * sizeof(type_of_data));
	hipMalloc((void**) &g_grad, core_length * sizeof(type_of_data));
	hipMemset(g_sum, 0, sum_size * core_length * sizeof(type_of_data));
	hipMemset(g_grad, 0, core_length * sizeof(type_of_data));

	int data_per_part = nnz / data_part + 1;

	if (model == 0) {

		for (int i = 0; i < data_part - 1; i++) {
			Update_Parameter_G_Gradient<<<grid_size, block_size,
			core_length * (block_size / core_dimen) * sizeof(type_of_data)>>>(
					order, core_length, core_dimen, parameter_a, parameter_g,
					data_per_part, value[i], index[i], g_sum);
			hipDeviceSynchronize();
		}
		Update_Parameter_G_Gradient<<<grid_size, block_size,
		core_length * (block_size / core_dimen) * sizeof(type_of_data)>>>(
				order, core_length, core_dimen, parameter_a, parameter_g,
				nnz - (data_part - 1) * data_per_part, value[data_part - 1],
				index[data_part - 1], g_sum);
		hipDeviceSynchronize();
	} else if (model == 1) {

		type_of_data *h_shared;
		hipMalloc((void**) &h_shared, sm * core_length * sizeof(type_of_data));
		hipMemset(h_shared, 0, sm * core_length * sizeof(type_of_data));

		for (int i = 0; i < data_part - 1; i++) {
			Update_Parameter_G_Gradient_Gobal<<<grid_size, block_size>>>(order,
					core_length, core_dimen, parameter_a, parameter_g,
					data_per_part, value[i], index[i], g_sum, h_shared);
			hipDeviceSynchronize();
		}
		Update_Parameter_G_Gradient_Gobal<<<
		grid_size, block_size>>>(order, core_length, core_dimen, parameter_a,
				parameter_g, nnz - (data_part - 1) * data_per_part,
				value[data_part - 1], index[data_part - 1], g_sum, h_shared);
		hipDeviceSynchronize();
		hipFree(h_shared);
	}

	Parameter_G_Gradient_Sum<<<core_length / (block_size / core_dimen) + 1,
	block_size>>>(order, core_length, core_dimen, nnz, g_sum, g_grad);
	hipDeviceSynchronize();
	Update_Parameter_G<<<core_length / (block_size / core_dimen) + 1,
	block_size>>>(order, core_length, core_dimen, parameter_g, g_grad,
			learn_rate_g, lambda_g);
	hipDeviceSynchronize();

	hipFree(g_sum);
	hipFree(g_grad);

}

__global__ void RMSE_AND_MAE(const int order, const int core_length,
		const int core_dimen, type_of_data **parameter_a, const int nnz,
		const type_of_data *value, const int *index, type_of_data **g,
		type_of_data *rmse, type_of_data *mae) {

	int core = core_dimen;
	int worker = block_size / core;
	int local = error_size / core;
	int lane_id = threadIdx.x % core;
	int local_id = threadIdx.x / core;
	int worker_id = worker * blockIdx.x + local_id;
	int workers = worker * gridDim.x;
	for (int nnz_index = worker_id; nnz_index < nnz; nnz_index += workers) {
		int order_index = nnz_index * order;
		type_of_data gs = 0.0;
		for (int g_index = 0; g_index < core_length / core_dimen; g_index++) {
			int parameter_a_index = g_index;
			type_of_data s = 1.0;
			for (int inner_order_index = 0; inner_order_index < order;
					inner_order_index++) {
				if (inner_order_index != 0) {
					s *= parameter_a[inner_order_index][index[order_index
							+ inner_order_index] * core_dimen
							+ parameter_a_index % core_dimen];
					parameter_a_index /= core_dimen;
				}

			}
			gs += s * g[0][g_index * core_dimen + lane_id];
		}

		type_of_data p_a_temp = parameter_a[0][index[order_index] * core_dimen
				+ lane_id];

		type_of_data p_a_gs = p_a_temp * gs;

		int temp = core;
		while (temp != 1) {
			temp /= 2;
			p_a_gs += __shfl_down_sync(mask, p_a_gs, temp);
		}

		p_a_gs = __shfl_sync(mask, p_a_gs, (local_id % local) * core);
		p_a_gs -= value[nnz_index];

		if (lane_id == 0) {
			atomicAdd(&rmse[nnz_index % error_size], p_a_gs * p_a_gs);
			atomicAdd(&mae[nnz_index % error_size], abs(p_a_gs));
		}

	}
}

void GET_RMSE_AND_MAE(const int order, const int core_length,
		const int core_dimen, type_of_data **parameter_a,
		type_of_data *parameter_g, const int nnz, const type_of_data *value,
		const int *index, type_of_data **g, type_of_data *rmse,
		type_of_data *mae) {

	Structure_Core_Tensor<<<core_length / (block_size / core_dimen) + 1,
	block_size>>>(order, core_length, core_dimen, parameter_g, g);
	hipDeviceSynchronize();

	type_of_data *errors_rmse;
	type_of_data *errors_mae;
	hipblasHandle_t handle_rmse;
	hipblasCreate(&handle_rmse);
	hipblasHandle_t handle_mae;
	hipblasCreate(&handle_mae);
	hipMalloc((void**) &errors_rmse, error_size * sizeof(type_of_data));
	hipMalloc((void**) &errors_mae, error_size * sizeof(type_of_data));
	hipMemset(errors_rmse, 0, error_size * sizeof(type_of_data));
	hipMemset(errors_mae, 0, error_size * sizeof(type_of_data));

	RMSE_AND_MAE<<<nnz / (block_size / core_dimen) + 1, block_size>>>(order,
			core_length, core_dimen, parameter_a, nnz, value, index, g,
			errors_rmse, errors_mae);
	hipDeviceSynchronize();

	type_of_data *rmse_sum = (type_of_data*) malloc(sizeof(type_of_data));
	type_of_data *mae_sum = (type_of_data*) malloc(sizeof(type_of_data));

	hipblasSasum(handle_rmse, error_size, errors_rmse, 1, rmse_sum);
	hipDeviceSynchronize();
	hipblasSasum(handle_mae, error_size, errors_mae, 1, mae_sum);
	hipDeviceSynchronize();

	*rmse = sqrt((*rmse_sum) / nnz);
	*mae = (*mae_sum) / nnz;
	hipFree(errors_rmse);
	hipFree(errors_mae);
	hipblasDestroy(handle_rmse);
	hipblasDestroy(handle_mae);
	free(rmse_sum);
	free(mae_sum);

}

void GET_RMSE_AND_MAE(const int order, const int core_length,
		const int core_dimen, type_of_data **parameter_a,
		type_of_data *parameter_g, const int nnz, type_of_data **value,
		int **index, type_of_data **g, type_of_data *rmse,
		type_of_data *mae) {

	Structure_Core_Tensor<<<core_length / (block_size / core_dimen) + 1,
	block_size>>>(order, core_length, core_dimen, parameter_g, g);
	hipDeviceSynchronize();

	type_of_data *errors_rmse;
	type_of_data *errors_mae;
	hipblasHandle_t handle_rmse;
	hipblasCreate(&handle_rmse);
	hipblasHandle_t handle_mae;
	hipblasCreate(&handle_mae);
	hipMalloc((void**) &errors_rmse, error_size * sizeof(type_of_data));
	hipMalloc((void**) &errors_mae, error_size * sizeof(type_of_data));
	hipMemset(errors_rmse, 0, error_size * sizeof(type_of_data));
	hipMemset(errors_mae, 0, error_size * sizeof(type_of_data));

	int data_per_part = nnz / data_part + 1;
	for (int i = 0; i < data_part - 1; i++) {
		RMSE_AND_MAE<<<data_per_part / (block_size / core_dimen) + 1,
		block_size>>>(order, core_length, core_dimen, parameter_a,
				data_per_part, value[i], index[i], g, errors_rmse, errors_mae);
		hipDeviceSynchronize();
	}

	RMSE_AND_MAE<<<data_per_part / (block_size / core_dimen) + 1, block_size>>>(
			order, core_length, core_dimen, parameter_a,
			nnz - (data_part - 1) * data_per_part, value[data_part - 1],
			index[data_part - 1], g, errors_rmse, errors_mae);
	hipDeviceSynchronize();

	type_of_data *rmse_sum = (type_of_data*) malloc(sizeof(type_of_data));
	type_of_data *mae_sum = (type_of_data*) malloc(sizeof(type_of_data));

	hipblasSasum(handle_rmse, error_size, errors_rmse, 1, rmse_sum);
	hipDeviceSynchronize();
	hipblasSasum(handle_mae, error_size, errors_mae, 1, mae_sum);
	hipDeviceSynchronize();

	*rmse = sqrt((*rmse_sum) / nnz);
	*mae = (*mae_sum) / nnz;
	hipFree(errors_rmse);
	hipFree(errors_mae);
	hipblasDestroy(handle_rmse);
	hipblasDestroy(handle_mae);
	free(rmse_sum);
	free(mae_sum);

}
