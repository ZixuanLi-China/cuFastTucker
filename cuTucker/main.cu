#include "hip/hip_runtime.h"
#include <stdio.h>
#include "tools.h"
#include "kernel.h"

//using namespace std;

type_of_data learn_alpha_a;
type_of_data learn_beta_a;
type_of_data lambda_a;

type_of_data learn_alpha_g;
type_of_data learn_beta_g;
type_of_data lambda_g;

int iter_number;

type_of_data learn_rate_a;
type_of_data learn_rate_g;

type_of_data train_rmse;
type_of_data train_mae;

type_of_data test_rmse;
type_of_data test_mae;

type_of_data best_train_rmse;
type_of_data best_train_mae;

type_of_data best_test_rmse;
type_of_data best_test_mae;

char *InputPath_train;
char *InputPath_test;

int order;
int core_length;

int *dimen;
int core_dimen;

double data_norm;

type_of_data rmse_train;
type_of_data rmse_test;

int nnz_train;
type_of_data **value_train_host;
type_of_data **value_train_device;
type_of_data **value_train_host_to_device;
int **index_train_host;
int **index_train_device;
int **index_train_host_to_device;

int nnz_test;
type_of_data *value_test_host;
type_of_data *value_test_device;
int *index_test_host;
int *index_test_device;

type_of_data **parameter_a_host;
type_of_data **parameter_a_host_to_device;
type_of_data **parameter_a_device;

type_of_data *parameter_g_host;
type_of_data *parameter_g_device;

type_of_data **g_device;
type_of_data **g_host_to_device;

double time_spend = 0.0;
double start_time;
double mid_time;
double stop_time;

int main(int argc, char *argv[]) {

	if (argc == 12) {

		InputPath_train = argv[1];
		InputPath_test = argv[2];

		order = atoi(argv[3]);
		core_dimen = atoi(argv[4]);

		iter_number = atoi(argv[5]);

		learn_alpha_a = atof(argv[6]);
		learn_beta_a = atof(argv[7]);
		lambda_a = atof(argv[8]);

		learn_alpha_g = atof(argv[9]);
		learn_beta_g = atof(argv[10]);
		lambda_g = atof(argv[11]);

		core_length = 1;
		for (int i = 0; i < order; i++) {
			core_length *= core_dimen;
		}

	}
	printf("learn_alpha_a:%f\tlearn_beta_a:%f\tlambda_a:%f\n", learn_alpha_a,
			learn_beta_a, lambda_a);
	printf("learn_alpha_g:%f\tlearn_beta_g:%f\tlambda_g:%f\n", learn_alpha_g,
			learn_beta_g, lambda_g);
	Getting_Input(InputPath_train, InputPath_test, order, &dimen, &nnz_train,
			&nnz_test, &index_train_host, &value_train_host, &index_test_host,
			&value_test_host, &data_norm);
	printf("nnz_train:\t%d\n", nnz_train);
	printf("nnz_test:\t%d\n", nnz_test);
	for (int i = 0; i < order; i++) {
		printf("order %d:\t%d\n", i + 1, dimen[i]);
	}
	printf("data_norm:\t%f\n", data_norm);

	Parameter_Initialization(order, core_length, core_dimen, dimen, data_norm,
			&parameter_a_host, &parameter_g_host);

	Cuda_Parameter_Initialization(order, core_length, core_dimen, dimen,
			nnz_train, value_train_host, &value_train_device,
			&value_train_host_to_device, index_train_host, &index_train_device,
			&index_train_host_to_device, nnz_test, value_test_host,
			&value_test_device, index_test_host, &index_test_device,
			parameter_a_host, parameter_g_host, &parameter_a_device,
			&parameter_g_device, &parameter_a_host_to_device, &g_device,
			&g_host_to_device);

	GET_RMSE_AND_MAE(order, core_length, core_dimen, parameter_a_device,
			parameter_g_device, nnz_train, value_train_host_to_device,
			index_train_host_to_device, g_device, &best_train_rmse,
			&best_train_mae);

	GET_RMSE_AND_MAE(order, core_length, core_dimen, parameter_a_device,
			parameter_g_device, nnz_test, value_test_device, index_test_device,
			g_device, &best_test_rmse, &best_test_mae);

	printf(
			"initial:\ttrain rmse:%f\ttest rmse:%f\ttrain mae:%f\ttest mae:%f\t\n",
			best_train_rmse, best_test_rmse, best_train_mae, best_test_mae);
	printf(
			"iter\ttrain rmse\ttest rmse\ttrain mae\ttest mae\tfactor time\tcore time\ttotal time\tcumulative time\n");

	for (int iter = 0; iter < iter_number; iter++) {

		learn_rate_a = learn_alpha_a / (1 + learn_beta_a * pow(iter, 1.5));
		learn_rate_g = learn_alpha_g / (1 + learn_beta_g * pow(iter, 1.5));

		start_time = Seconds();

		Update_Parameter_A(order, core_length, core_dimen, parameter_a_device,
				nnz_train, value_train_host_to_device,
				index_train_host_to_device, g_device, learn_rate_a, lambda_a);

		mid_time = Seconds();

		Update_Parameter_G_Batch(order, core_length, core_dimen,
				parameter_a_device, parameter_g_device, nnz_train,
				value_train_host_to_device, index_train_host_to_device,
				learn_rate_g, lambda_g);

		stop_time = Seconds();
		time_spend += stop_time - start_time;

		GET_RMSE_AND_MAE(order, core_length, core_dimen, parameter_a_device,
				parameter_g_device, nnz_train, value_train_host_to_device,
				index_train_host_to_device, g_device, &train_rmse, &train_mae);

		GET_RMSE_AND_MAE(order, core_length, core_dimen, parameter_a_device,
				parameter_g_device, nnz_test, value_test_device,
				index_test_device, g_device, &test_rmse, &test_mae);

		Select_Best_Result(&train_rmse, &train_mae, &test_rmse, &test_mae,
				&best_train_rmse, &best_train_mae, &best_test_rmse,
				&best_test_mae);
		printf("%d\t%f\t%f\t%f\t%f\t%f\t%f\t%f\t%f\n", iter, train_rmse,
				test_rmse, train_mae, test_mae, mid_time - start_time,
				stop_time - mid_time, stop_time - start_time, time_spend);

	}

	printf("best:\ttrain rmse:%f\ttest rmse:%f\ttrain mae:%f\ttest mae:%f\t\n",
			best_train_rmse, best_test_rmse, best_train_mae, best_test_mae);

	return 0;
}
